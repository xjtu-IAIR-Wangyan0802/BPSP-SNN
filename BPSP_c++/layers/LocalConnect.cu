#include "hip/hip_runtime.h"
#include "LocalConnect.h"
#include "../common/cuBase.h"
#include "../common/Config.h"
#include "../layers/BranchLayer.h"



/*
dim3 block = dim3(batch, outputAmount);
dim3 thread= min(outputDim * outputDim, 512);
*/
__global__ void g_LocalConnect_backpropagation_kernelSize1(
	float* _curDelta,
	float**_w,
	float* _nextDelta,
	int     dim,
	int     area,
	int localKernelSize);

/*
 * block = dim3(outputAmount, kernelSize * kernelSize);
 * thread= dim3(batch);
*/
__global__ void g_LocalConnect_wgrad_Add(
	float** _WgradTmp,
	float** Wgrad,
	float** w,
	int kernelSize,
	int batch,
	float lambda,
	int wgradTmpArea,
	int wgradArea,
	int wArea);

/*
dim3 block = dim3(batch, outputAmount);
dim3 thread= min(16, min(outputDim * outputDim, 64));
*/
__global__ void g_LocalConnect_wgrad_kernelSize1(
	float* _inputs,
	float* _curDelta,
	float** _wgradTmp,
	/*float** _w,*/
	int dim,
	int area,
	int batch,
	float lambda);
/*
 *dim3 block = dim3(batch, amount);
 *dim3 thread= dim3(16, min(outputDim * outputDim, 64));
*/

__global__ void g_LocalConnect_feedforward_1(
	float** arrayS,
	float** arrayW,
	float** arrayB,
	float* _output,
	int inputSize,
	int kernelSize,
	int outputDim,
	int outputArea,
	int batch,
	int k1Amount,
	int localKernelSize);

template <int OUTPUTDIM2, int THREADS>
__global__ void g_LocalConnect_feedforward_s_2(
	float*  inputs,
	float** arrayW,
	float** arrayB,
	float* _output,
	int inputSize,
	int kernelSize,
	int outputSize,
	int inputArea,
	int outputArea,
	int batch,
	int k1Amount,
	int localKernelSize);

/*
 * function: get convolution layer and pooling output
 * dim3 block = dim3(batch, amount);
 * dim3 thread= dim3(min(outputDim * outputDim, 512));
 * const kernelsize = 1
*/

__global__ void g_LocalConnect_feedforward_kernelSize1_2(
	float*  inputs,
	float** arrayW,
	float** arrayB,
	float* _output,
	int dim,
	int area,
	int batch,
	int k1Amount,
	int localKernelSize);

/*
dim3 block = dim3(batch, outputAmount);
dim3 thread= min(outputDim * outputDim, 512);
*/
__global__ void g_LocalConnect_backpropagation(
	float* _convDelta,
	float**_w,
	float* _poolDelta,
	int     _convOutputSize,
	int     _poolOutputSize,
	int     _kernelAmount1,
	int     _kernelAmount2,
	int     _kernelSize,
	int     _convDeltaArea,
	int     _poolDeltaArea,
	int localKernelSize);
/*
 *function: get convolution layer and pooling output
 *dim3 block = dim3(batch, amount);
 *dim3 thread= dim3(min(outputDim * outputDim, 256));
*/

__global__ void g_LocalConnect_feedforward_2(
	float*  inputs,
	float** arrayW,
	float** arrayB,
	float* _output,
	int inputSize,
	int kernelSize,
	int outputSize,
	int inputArea,
	int outputArea,
	int batch,
	int k1Amount,
	int localKernelSize);

/*
* blocks  : dim3(batch, cuKernelScan[cl] * localKernelSize, Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_LocalConnect_wgrad(
	float* _inputs,
	float* _curDelta,
	float** _wgrad,
	int inputDim,
	int curDeltaDim,
	int kernelSize,
	int inputAmount,
	int outputAmount,
	int inputArea,
	int curDeltaAea,
	int batch,
	float lambda);

/*
* blocks  : dim3(batch, cuKernelScan[cl], Config::instance()->getChannels()),
* threads : dim3(threadidx)
*/
__global__ void g_LocalConnect_wgrad_1(float** sArray,
	float* convDelta,
	float* WgradTmp,
	int imgSize,
	int convOutputSize,
	int kernelAmount2,
	int kernelSize,
	int sArrayArea,
	int convDeltaArea,
	int wgrapTmpArea,
	int localKernelSize);

/*
 *block = dim3(localKernelSize, amount);
 *thread= dim3(batch);
 *
*/
__global__ void g_LocalConnect_Bgrad(float* delta,
	float** bgrad,
	int deltaSize,
	int kernelAmount2,
	int batch,
	int deltaArea,
	int localKernelSize);


void LocalConnect::calCost()
{
	cost->gpuClear();
	g_getCost_3<<<dim3(w.size()), dim3(32), sizeof(float) * 32>>>(cost->getDev(), 
		w.m_devPoint, 
		lambda,
		w[0]->getLen());
	hipStreamSynchronize(0);
	getLastCudaError("LocalConnect:getCost");
}

void LocalConnect::feedforward()
{
	if((kernelSize == 3 || kernelSize == 5) && inputDim >= 4 && inputDim <= 8){
		dim3 block = dim3(batch, outputAmount);
		const int threads = 8;
		dim3 thread= dim3(threads, outputDim * outputDim);
		if(outputDim == 4){
			g_LocalConnect_feedforward_s_2<16, threads><<<block, thread>>>(inputs->getDev(), w.m_devPoint, b.m_devPoint, outputs->getDev(), inputDim,
				kernelSize, outputDim, inputs->getArea(), outputs->getArea(), batch, outputAmount, localKernelSize);
		}else if(outputDim == 5){
			g_LocalConnect_feedforward_s_2<25, threads><<<block, thread>>>(inputs->getDev(), w.m_devPoint, b.m_devPoint, outputs->getDev(), inputDim,
				kernelSize, outputDim, inputs->getArea(), outputs->getArea(), batch, outputAmount, localKernelSize);
		}else if(outputDim == 6){
			g_LocalConnect_feedforward_s_2<36, threads><<<block, thread>>>(inputs->getDev(), w.m_devPoint, b.m_devPoint, outputs->getDev(), inputDim,
				kernelSize, outputDim, inputs->getArea(), outputs->getArea(), batch, outputAmount, localKernelSize);
		}else if(outputDim == 7){
			g_LocalConnect_feedforward_s_2<49, threads><<<block, thread>>>(inputs->getDev(), w.m_devPoint, b.m_devPoint, outputs->getDev(), inputDim,
				kernelSize, outputDim, inputs->getArea(), outputs->getArea(), batch, outputAmount, localKernelSize);
		}else if(outputDim == 8){
			g_LocalConnect_feedforward_s_2<64, threads><<<block, thread>>>(inputs->getDev(), w.m_devPoint,  b.m_devPoint, outputs->getDev(), inputDim,
				kernelSize, outputDim, inputs->getArea(), outputs->getArea(), batch, outputAmount, localKernelSize);
		}

		checkCudaErrors(hipStreamSynchronize(0));
		getLastCudaError("LocalConnect:g_LocalConnect_feedforward_s_2");
	}
	else if(kernelSize == 1){
		dim3 block = dim3(batch, outputAmount);
		dim3 thread= dim3(min(outputDim * outputDim, 512));

		g_LocalConnect_feedforward_kernelSize1_2<<<block, thread>>>(
			inputs->getDev(),
			w.m_devPoint, 
			b.m_devPoint,
			outputs->getDev(),
			inputDim,
			inputs->getArea(),
			batch,
			outputAmount,
			localKernelSize);
		checkCudaErrors(hipStreamSynchronize(0));
		getLastCudaError("LocalConnect:g_LocalConnect_feedforward_kernelSize1_2");
	}
	else {
		dim3 block = dim3(batch, outputAmount);
		dim3 thread= dim3(8, min(outputDim * outputDim, 64));
		g_LocalConnect_feedforward_2<<<block, thread,
			sizeof(float) * outputDim * outputDim>>>
			(inputs->getDev(),
			w.m_devPoint, 
			b.m_devPoint,
			outputs->getDev(),
			inputDim,
			kernelSize,
			outputDim,
			inputs->getArea(),
			outputs->getArea(),
			batch,
			outputAmount,
			localKernelSize);
		checkCudaErrors(hipStreamSynchronize(0));
		getLastCudaError("LocalConnect:g_LocalConnect_feedforward_2");
	}

	if(NON_LINEARITY >= 0){
		dim3 thread = dim3(min(256, outputs->getLen()));
		dim3 block  = dim3(min(256, (outputs->getLen() + thread.x - 1) / thread.x));
		g_nonLinearity<<<block, thread>>>(
			outputs->getDev(), 
			outputs->getLen(),
			NON_LINEARITY);
		checkCudaErrors(hipStreamSynchronize(0));
		getLastCudaError("LocalConnect::g_nonLinearity");
	}
}

void LocalConnect::backpropagation()
{
	if(NON_LINEARITY >= 0){
		dim3 thread = dim3(min(256, outputs->getLen()));
		dim3 block  = dim3(min(256, (outputs->getLen() + thread.x - 1) / thread.x));

		g_dnonLinearity<<<block, thread>>>(curDelta->getDev(),
			outputs->getDev(), curDelta->getLen(), NON_LINEARITY);

		checkCudaErrors(hipStreamSynchronize(0));
		getLastCudaError("LocalConnect::g_dnonLinearity");
	}
	
	if(inputs){
		dim3 block = dim3(batch, outputAmount);
		dim3 thread= dim3(min(outputDim * outputDim, 512));

		preDelta->gpuClear();

 		if(kernelSize == 1){
 			g_LocalConnect_backpropagation_kernelSize1<<<block, thread>>>(
 				curDelta->getDev(),
 				w.m_devPoint,
 				preDelta->getDev(),
 				outputDim,
 				curDelta->getArea(),
 				localKernelSize);
 			checkCudaErrors(hipStreamSynchronize(0));
 			getLastCudaError("LocalConnect::g_LocalConnect_backpropagation_kernelSize1");
 
 		}else{
			g_LocalConnect_backpropagation<<<block, thread>>>(
				curDelta->getDev(),
				w.m_devPoint,
				preDelta->getDev(),
				outputDim,
				inputDim,
				inputAmount,
				outputAmount,
				kernelSize,
				curDelta->getArea(),
				preDelta->getArea(),
				localKernelSize);
			checkCudaErrors(hipStreamSynchronize(0));
			getLastCudaError("LocalConnect::g_LocalConnect_backpropagation");
		}
	}
}


void LocalConnect::getGrad()
{
	if(kernelSize == 1){
		dim3 block = dim3(batch, outputAmount);
		dim3 thread= dim3(min(outputDim * outputDim, 512));
		g_LocalConnect_wgrad_kernelSize1<<<block, thread, sizeof(float) * batch>>>(
			inputs->getDev(),
			curDelta->getDev(),
			wgradTmp.m_devPoint,
			inputDim,
			inputs->getArea(),
			batch,
			lambda);
		checkCudaErrors(hipStreamSynchronize(0));
		getLastCudaError("g_LocalConnect_wgrad_kernelSize1");

		block  = dim3(outputAmount, kernelSize * kernelSize);
		thread = dim3(batch);
	}
	else{
		dim3 block = dim3(batch, outputAmount);
		dim3 thread= min(9, min(outputDim * outputDim, 64));
		g_LocalConnect_wgrad<<<block, thread, sizeof(float) * inputDim * inputDim>>>(
			inputs->getDev(),
			curDelta->getDev(),
			wgradTmp.m_devPoint,
			inputDim,
			outputDim,
			kernelSize,
			inputAmount,
			outputAmount,
			inputs->getArea(),
			curDelta->getArea(),
			batch,
			lambda);

		checkCudaErrors(hipStreamSynchronize(0));
		getLastCudaError("g_LocalConnect_wgrad");
	}

	dim3 block  = dim3(outputAmount * localKernelSize, kernelSize * kernelSize);
	dim3 thread = dim3(batch);
	g_LocalConnect_wgrad_Add<<<block, thread, sizeof(float) * batch>>>(
		wgradTmp.m_devPoint,
		wgrad.m_devPoint,
		w.m_devPoint,
		kernelSize,
		batch,
		lambda,
		wgradTmp[0]->getArea(),
		wgrad[0]->getArea(),
		w[0]->getArea());
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("g_LocalConnect_wgrad_Add");

	block = dim3(localKernelSize, outputAmount);
	thread= dim3(batch);
	g_LocalConnect_Bgrad<<<block,thread,sizeof(float) * batch>>>
		(curDelta->getDev(),
		bgrad.m_devPoint,
		outputDim,
		outputAmount,
		batch,
		curDelta->getArea(),
		localKernelSize);

	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("LocalConnect::getGrad::g_LocalConnect_Bgrad");
}

void LocalConnect::updateWeight()
{
	dim3 thread = min(256, w[0]->getLen());
	dim3 block  = momentum_w.size();
	g_vecAdd<<<block, thread, 0, Layers::instance()->get_stream()>>>(momentum_w.m_devPoint, wgrad.m_devPoint, w.m_devPoint,
		momentum_b.m_devPoint, bgrad.m_devPoint, b.m_devPoint,
		w[0]->getLen(), b[0]->getLen(), 
		Config::instance()->getMomentum(),
		Config::instance()->getLrate(), Config::instance()->getLrate());
}

LocalConnect::LocalConnect(std::string name)
{
	m_name = name;
	ConfigLocal* config = static_cast<ConfigLocal*>(Config::instance()->getLayerByName(m_name));
	ConvLayerBase * preLayer = (ConvLayerBase*)Layers::instance()->get(config->m_input);

	inputs = preLayer->getOutputs();
	if(inputs == NULL){
		/*inputs = NULL the type must be BranchLayers*/
		Assert(Config::instance()->getLayerByName(config->m_input)->isBranchLayer());
		Assert(config->m_subInput != std::string("NULL"));
		BranchLayer* bl = static_cast<BranchLayer*>(preLayer);
		inputs = bl->getSubOutput(config->m_subInput);
		preDelta = bl->getSubCurDelta(config->m_subInput);
	}else{
		preDelta = preLayer->getCurDelta();
	}

	inputAmount = preLayer->outputAmount;
	outputAmount = inputAmount;
	kernelSize = config->m_kernelSize;

	inputDim  = preLayer->outputDim;
	outputDim = inputDim;
	batch     = Config::instance()->getBatchSize();
	lambda    = config->m_weightDecay;
	NON_LINEARITY = config->m_nonLinearity;

	localKernelSize = outputDim * outputDim;
	outputs = new cuMatrix<float> (batch, outputDim * outputDim, outputAmount);
	curDelta = new cuMatrix<float>(batch, outputDim * outputDim, outputAmount);

	for(int i = 0; i < outputAmount * localKernelSize; i++){
		w.push_back(new cuMatrix<float>(kernelSize, kernelSize, 1));
		b.push_back(new cuMatrix<float>(1, 1, 1));
		wgrad.push_back(new cuMatrix<float>(kernelSize, kernelSize, 1));
		bgrad.push_back(new cuMatrix<float>(1, 1, 1));
		wgradTmp.push_back(new cuMatrix<float>(batch, kernelSize * kernelSize, 1));
	}

	w.toGpu();
	b.toGpu();
	wgrad.toGpu();
	bgrad.toGpu();
	wgradTmp.toGpu();

	for(int i = 0; i < outputAmount * localKernelSize; i++){
		momentum_w.push_back(new cuMatrix<float>(kernelSize, kernelSize, 1));
		momentum_b.push_back(new cuMatrix<float>(1, 1, 1));
	}
	momentum_w.toGpu();
	momentum_b.toGpu();

	this->initRandom();
	Layers::instance()->set(m_name, this);
}

void LocalConnect::save(FILE* file)
{
	for(int a = 0; a < (int)w.size(); a++){
		w[a]->toCpu();
		b[a]->toCpu();
		for(int c = 0; c < w[a]->channels; c++){
			for(int i = 0; i < w[a]->rows; i++){
				for(int j = 0; j < w[a]->cols; j++){
					fprintf(file, "%f ", w[a]->get(i, j, c));
				}
			}
		}

		for(int c = 0; c < b[a]->channels; c++){
			for(int i = 0; i < b[a]->rows; i++){
				for(int j = 0; j < b[a]->cols; j++){
					fprintf(file, "%f ", b[a]->get(i, j, c));
				}
			}
		}
	}
}

void LocalConnect::clearMomentum()
{
	for(int i = 0; i < (int)momentum_b.size(); i++){
		momentum_b[i]->gpuClear();
	}
	for(int i = 0; i < (int)momentum_w.size(); i++){
		momentum_w[i]->gpuClear();
	}
}

void LocalConnect::initRandom()
{
	//srand(clock());
	float initW = Config::instance()->getLayerByName(m_name)->m_initW;

	if(Config::instance()->getLayerByName(m_name)->isGaussian()){
		for(int i = 0; i < (int)w.size(); i++){
			float epsilon = initW;
			for(int c = 0; c < w[i]->channels; c++)
			{
				float r1 = 0.01f + 5.0f * (rand()) / RAND_MAX;
				float r2 = 0.01f + 5.0f * (rand()) / RAND_MAX;
				createGaussian(w[i]->getHost() + c * w[i]->getArea(), r1,r2,
					kernelSize, kernelSize, w[i]->channels,
					epsilon);
			}
			w[i]->toGpu();
		}
	}
	else{
		for(int i = 0; i < (int)w.size(); i++){
			for(int j = 0; j < w[i]->getLen(); j++){
				w[i]->getHost()[j] =  initW * (2.0f * rand() / RAND_MAX - 1.0f);
				//printf("%f ", w[i]->hostData[j]);
			}//printf("\n");
			w[i]->toGpu();
		}
	}
}

void LocalConnect::initFromCheckpoint(FILE* file)
{
	float val = 0;
	for(int a = 0; a < (int)w.size(); a++){
		for(int c = 0; c < w[a]->channels; c++){
			for(int i = 0; i < w[a]->rows; i++){
				for(int j = 0; j < w[a]->cols; j++){
					if(fscanf(file, "%f", &val) == EOF){
                        LOG("scanf fail", "result/log.txt");
                    }
					w[a]->set(i, j, c, val);
				}
			}
		}

		for(int c = 0; c < b[a]->channels; c++){
			for(int i = 0; i < w[a]->rows; i++){
				for(int j = 0; j < w[a]->cols; j++){
					if(fscanf(file, "%f", &val) == EOF){
                        LOG("scanf fail", "result/log.txt");
                    }
					b[a]->set(i, j, c, val);
				}
			}
		}
		w[a]->toGpu();
		b[a]->toGpu();
	}
}

/*
 *dim3 block = dim3(batch, amount);
 *dim3 thread= dim3(16, min(outputDim * outputDim, 64));
*/

__global__ void g_LocalConnect_feedforward_1(
	float** arrayS,
	float** arrayW,
	float** arrayB,
	float* _output,
	int inputSize,
	int kernelSize,
	int outputDim,
	int outputArea,
	int batch,
	int k1Amount,
	int localKernelSize)
{
	extern __shared__ float image[];
	int sp = blockIdx.x;
	int k  = blockIdx.y;

	int OutputSize2 = outputDim  * outputDim;
	int inputSize2  = inputSize  * inputSize;
	int kernelSize2 = kernelSize * kernelSize;

	float* curInput  = arrayS[sp] + k * inputSize2;
	float* curOutput = _output + outputArea * k + sp * OutputSize2;

	/*load the image to shared memory*/
	for(int i = 0; i < inputSize2; i += blockDim.x * blockDim.y){
		int id = i + threadIdx.x + threadIdx.y * blockDim.x;
		if(id < inputSize2){
			image[id] = curInput[id];
		}
	}
	__syncthreads();

	int padding = kernelSize >> 1;
	/*convolution*/
	for(int ty = 0; ty < OutputSize2; ty += blockDim.y)
	{
		int tyid = ty + threadIdx.y;
		if(tyid < OutputSize2)
		{
			int x = tyid / outputDim;
			int y = tyid % outputDim;
			float val = 0.0;
			float* w        = arrayW[k * localKernelSize + tyid];
			float  b        = arrayB[k * localKernelSize + tyid][0];

			for(int tx = 0; tx < kernelSize2; tx += blockDim.x){
				int txid = tx + threadIdx.x;
				if(txid < kernelSize2){
					int i = txid / kernelSize;
					int j = txid % kernelSize;
					int xx = x + i - padding;
					int yy = y + j - padding;
					if(xx >= 0 && xx < inputSize && yy >= 0 && yy < inputSize)
						val += image[xx * inputSize + yy] * w[i * kernelSize + j];
				}
			}
			curOutput[tyid] = val + b;
		}
	}
}


/*
 * function: get convolution layer and pooling output
 * dim3 block = dim3(batch, amount);
 * dim3 thread= dim3(8, min(outputDim * outputDim, 64));
*/

__global__ void g_LocalConnect_feedforward_2(
	float*  inputs,
	float** arrayW,
	float** arrayB,
	float* _output,
	int inputSize,
	int kernelSize,
	int outputSize,
	int inputArea,
	int outputArea,
	int batch,
	int k1Amount,
	int localKernelSize)
{
	extern __shared__ float image[];
	int sp = blockIdx.x;
	int k  = blockIdx.y;

	int outputSize2 = outputSize * outputSize;
	int inputSize2  = inputSize  * inputSize;
	int kernelSize2 = kernelSize * kernelSize;

	float* curInput  = inputs  + k * inputArea  + sp * inputSize2;
	float* curOutput = _output + k * outputArea + sp * outputSize2;

	/*load the image to shared memory*/
	for(int i = 0; i < inputSize2; i += blockDim.x * blockDim.y){
		int id = i + threadIdx.x + threadIdx.y * blockDim.x;
		if(id < inputSize2){
			image[id] = curInput[id];
			curOutput[id] = 0;
		}
	}
	__syncthreads();

	int padding = kernelSize >> 1;
	/*convolution*/
	for(int ty = 0; ty < outputSize2; ty += blockDim.y)
	{
		int tyid = ty + threadIdx.y;
		if(tyid < outputSize2)
		{
			int x = tyid / outputSize;
			int y = tyid % outputSize;
			float val = 0.0;
			float* w = arrayW[k * localKernelSize + tyid];

			for(int tx = 0; tx < kernelSize2; tx += blockDim.x){
				int txid = tx + threadIdx.x;
				if(txid < kernelSize2){
					int i = txid / kernelSize;
					int j = txid % kernelSize;
					int xx = x + i - padding;
					int yy = y + j - padding;
					if(xx >= 0 && xx < inputSize && yy >= 0 && yy < inputSize)
						val += image[xx * inputSize + yy] * w[i * kernelSize + j];
				}
			}
			atomicAdd(curOutput + tyid, val);
		}
	}

	__syncthreads();

	for(int i = 0; i < outputSize2; i += blockDim.y * blockDim.x)
	{
		int id = i + threadIdx.y * blockDim.x + threadIdx.x;
		if(id < outputSize2)
		{
			float  b = arrayB[k * localKernelSize + id][0];
			curOutput[id] += b;
		}
	}
}



/*
 * function: get convolution layer and pooling output
 * dim3 block = dim3(batch, amount);
 * dim3 thread= dim3(min(outputDim * outputDim, 512));
 * const kernelsize = 1
*/

__global__ void g_LocalConnect_feedforward_kernelSize1_2(
	float*  inputs,
	float** arrayW,
	float** arrayB,
	float* _output,
	int dim,
	int area,
	int batch,
	int k1Amount,
	int localKernelSize)
{
	int sp = blockIdx.x;
	int k  = blockIdx.y;

	int outputSize2 = dim * dim;
	int inputSize2  = dim * dim;

	float* curInput  = inputs  + k * area + sp * inputSize2;
	float* curOutput = _output + k * area + sp * outputSize2;

	/*convolution*/
	for(int ty = 0; ty < outputSize2; ty += blockDim.x)
	{
		int tyid = ty + threadIdx.x;
		if(tyid < outputSize2)
		{
			int skip = k * localKernelSize + tyid;
			float val = 0.0;
			float w = arrayW[skip][0];
			float b = arrayB[skip][0];
			val = curInput[tyid] * w + b;
			curOutput[tyid] = val ;
		}
	}
}


/*
 * function: get convolution layer and pooling output
 * dim3 block = dim3(batch, amount);
 * dim3 thread= dim3(8, min(outputDim * outputDim, 64));
 2<64, 9, 8, 8, 64>
*/
template <int OUTPUTDIM2, int THREADS>
__global__ void g_LocalConnect_feedforward_s_2(
	float*  inputs,
	float** arrayW,
	float** arrayB,
	float* _output,
	int inputSize,
	int kernelSize,
	int outputSize,
	int inputArea,
	int outputArea,
	int batch,
	int k1Amount,
	int localKernelSize)
{
	__shared__ float image[OUTPUTDIM2];
	int sp = blockIdx.x;
	int k  = blockIdx.y;
	__shared__ float convSum[OUTPUTDIM2][THREADS];

	int outputSize2 = outputSize * outputSize;
	int inputSize2  = inputSize  * inputSize;
	int kernelSize2 = kernelSize * kernelSize;

	float* curInput  = inputs  + k * inputArea  + sp * inputSize2;
	float* curOutput = _output + k * outputArea + sp * outputSize2;

	/*load the image to shared memory*/
	for(int i = 0; i < inputSize2; i += blockDim.x * blockDim.y){
		int id = i + threadIdx.x + threadIdx.y * blockDim.x;
		if(id < inputSize2){
			image[id] = curInput[id];
		}
	}
	__syncthreads();

	int padding = kernelSize >> 1;
	/*convolution*/

	for(int ty = 0; ty < outputSize2; ty += blockDim.y)
	{
		int tyid = ty + threadIdx.y;
		
		if(tyid < outputSize2)
		{
			int x = tyid / outputSize;
			int y = tyid % outputSize;
			float val = 0.0;
			float* w = arrayW[k * localKernelSize + tyid];
			float* _convSum = convSum[threadIdx.y];
			float  b = arrayB[k * localKernelSize + tyid][0];
			_convSum[threadIdx.x] = 0;
			

			for(int tx = 0; tx < kernelSize2; tx += blockDim.x){
				int txid = tx + threadIdx.x;
				if(txid < kernelSize2){
					int i = txid / kernelSize;
					int j = txid % kernelSize;
					int xx = x + i - padding;
					int yy = y + j - padding;
					if(xx >= 0 && xx < inputSize && yy >= 0 && yy < inputSize)
						val += image[xx * inputSize + yy] * w[i * kernelSize + j];
				}
			}
			_convSum[threadIdx.x] = val;
			__syncthreads();
#pragma  unroll
			for(int len = THREADS; len != 1; len = (len + 1) >> 1){
				int skip = (len + 1) >> 1;
				if(threadIdx.x < (len >> 1)) _convSum[threadIdx.x] += _convSum[threadIdx.x + skip];
				__syncthreads();
			}
			if(threadIdx.x == 0)
				curOutput[tyid] = _convSum[0] + b;
		}
	}
}


/*
dim3 block = dim3(batch, outputAmount);
dim3 thread= min(outputDim * outputDim, 512);
*/
__global__ void g_LocalConnect_backpropagation_kernelSize1(
	float* _curDelta,
	float**_w,
	float* _nextDelta,
	int     dim,
	int     area,
	int localKernelSize)
{
	int s = blockIdx.x;
	int k = blockIdx.y;

	int dim2 = dim * dim;
	int skip = k * area + s * dim2;
	float* curDelta = _curDelta  + skip;
	float* nxtDelta = _nextDelta + skip;

	for (int tidx = 0; tidx < dim2; tidx += blockDim.x) {
		int idx = tidx + threadIdx.x;
		if (idx < dim2) {
			float val = 0.0;
			float w = _w[k * localKernelSize + idx][0];
			val = curDelta[idx] * w;
			nxtDelta[idx] = val;
		}
	}
}


/*
dim3 block = dim3(batch, outputAmount);
dim3 thread= min(outputDim * outputDim, 512);
*/
__global__ void g_LocalConnect_backpropagation (
	float* _convDelta,
	float**_w,
	float* _poolDelta,
	int     _convOutputSize,
	int     _poolOutputSize,
	int     _kernelAmount1,
	int     _kernelAmount2,
	int     _kernelSize,
	int     _convDeltaArea,
	int     _poolDeltaArea,
	int localKernelSize)
{
	int curSize = _convOutputSize;
	int wSize = _kernelSize;
	int nxtSize = _poolOutputSize;

	int s = blockIdx.x;
	int k = blockIdx.y;

	int curSize2 = curSize * curSize;
	int nxtSize2 = nxtSize * nxtSize;
	float* curDelta = _convDelta + k * _convDeltaArea + s * curSize2;
	float* nxtDelta = _poolDelta + k * _poolDeltaArea + s * nxtSize2;

	int half = wSize >> 1;
	for (int tidx = 0; tidx < nxtSize2; tidx += blockDim.x) {
		int idx = tidx + threadIdx.x;
		if (idx < nxtSize2) {
			int i = idx / nxtSize;
			int j = idx % nxtSize;
			float val = 0.0;
			for (int x = 0; x < wSize; x++) {
				for (int y = 0; y < wSize; y++) {
					int cx = i + (half - x);
					int cy = j + (half - y);
					int wx = x;
					int wy = y;
					if(cx >= 0 && cx < curSize && cy >= 0 && cy < curSize){
						float* w = _w[k * localKernelSize + cx * curSize + cy];
						val += curDelta[cx * curSize + cy] * w[wx * wSize + wy];
					}
				}
			}
			nxtDelta[idx] = val;
		}
	}
}

/*
 * dim3 block = dim3(batch, outputAmount);
 * dim3 thread= dim3(min(outputDim * outputDim, 512));
*/
__global__ void g_LocalConnect_wgrad_kernelSize1(
	float* _inputs,
	float* _curDelta,
	float** _wgradTmp,
	int dim,
	int area,
	int batch,
	float lambda)
{
	int b  = blockIdx.x;
	int k  = blockIdx.y;

	int dim2 = dim * dim;

	int skip = k * area + b * dim2;
	float* input    = _inputs + skip;
	float* curDelta = _curDelta + skip;

	for(int y = 0; y < dim2; y += blockDim.x){
		int yid = y + threadIdx.x;
		if(yid < dim2){
			skip = k * dim2 + yid;
			float val = input[yid] * curDelta[yid];
			//_wgradTmp[skip][0] = val / batch + lambda * _w[skip][0];
			_wgradTmp[skip][0] = val;
		}
	}
}

/*
 *dim3 block = dim3(batch, outputAmount);
 *dim3 thread= min(9, min(outputDim * outputDim, 64));
*/
__global__ void g_LocalConnect_wgrad(
	float* _inputs,
	float* _curDelta,
	float** _wgradTmp,
	/*float** _w,*/
	int inputDim,
	int curDeltaDim,
	int kernelSize,
	int inputAmount,
	int outputAmount,
	int inputArea,
	int curDeltaAea,
	int batch,
	float lambda)
{
	int sp = blockIdx.x;
	int k  = blockIdx.y;

	extern __shared__ float image[];

	int inputSize2    = inputDim * inputDim;
	int curDeltaSize2 = curDeltaDim * curDeltaDim;
	int kernelSize2   = kernelSize * kernelSize;

	float* input = _inputs + k * inputArea + sp * inputSize2;


	/*load the image to shared memory*/
	for(int i = 0; i < inputSize2; i += blockDim.x * blockDim.y){
		int id = i + threadIdx.x + threadIdx.y * blockDim.x;
		if(id < inputSize2){
			image[id] = input[id];
		}
		
	}
	__syncthreads();


	float* curDelta = _curDelta + k * curDeltaAea + sp * curDeltaSize2;

	int half = (kernelSize >> 1);
	for(int y = 0; y < curDeltaSize2; y += blockDim.y){
		int yid = y + threadIdx.y;
		if(yid < curDeltaSize2){
			int ox = yid / curDeltaDim;
			int oy = yid % curDeltaDim;
			float* wgrad = _wgradTmp[k * curDeltaSize2 + yid] + sp * kernelSize2;
			float  delta = curDelta[yid];
			for(int x =  0; x < kernelSize2; x+= blockDim.x){
				int xid = x + threadIdx.x;
				if(xid < kernelSize2){
					int i = xid / kernelSize;
					int j = xid % kernelSize;
				
					int rox = ox + i - half;
					int roy = oy + j - half;
					if(rox >= 0 && rox < inputDim && roy >=0 && roy < inputDim){
						float val  = image[rox * inputDim + roy] * delta;
						wgrad[xid] = val;
					}else{
						wgrad[xid] = 0;
					}
				}
			}
		}
	}
}

/*
 *block = dim3(localKernelSize, amount)
 *thread= dim3(batch)
*/
__global__ void g_LocalConnect_Bgrad(float* _delta,
	float** bgrad,
	int deltaSize,
	int kernelAmount2,
	int batch,
	int deltaArea,
	int localKernelSize)
{
	extern __shared__ float _sum[];
	int local = blockIdx.x;
	int k     = blockIdx.y;
	int sp    = threadIdx.x;

	int deltaSize2 = deltaSize * deltaSize;
	float delta = _delta[k * deltaArea + sp * deltaSize2 + local];
	_sum[sp] = delta;
	__syncthreads();

	int len = batch;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		bgrad[k * localKernelSize + local][0] = _sum[0] / batch;
	}
}



/*
 * block = dim3(outputAmount, kernelSize * kernelSize);
 * thread= dim3(batch);
*/
__global__ void g_LocalConnect_wgrad_Add(
	float** _WgradTmp,
	float** Wgrad,
	float** w,
	int kernelSize,
	int batch,
	float lambda,
	int wgradTmpArea,
	int wgradArea,
	int wArea)
{
	extern __shared__ float _sum[];
	int ok  = blockIdx.x;
	int kid = blockIdx.y;
	int tid = threadIdx.x;
	_sum[threadIdx.x] = 0;
	__syncthreads();
	int tlen = batch;
	float* wgradTmp = _WgradTmp[ok];
	int kernelSize2 = kernelSize * kernelSize;
	for(int i = 0; i < tlen; i += blockDim.x)
	{
		int b = i + threadIdx.x;
		if(b < tlen)
		{
			_sum[threadIdx.x] += wgradTmp[b * kernelSize2 + kid];
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(tid < (len >> 1))
		{
			_sum[tid] += _sum[tid + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	if(tid == 0)
	{
		Wgrad[ok][kid] = _sum[0] / batch + w[ok][kid] * lambda;
	}
}
